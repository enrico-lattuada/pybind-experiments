#include "hip/hip_runtime.h"
#include "foo_gpu.cuh"

__global__ void add_kernel(double *a, double *b, double *c)
{
    *c = *a + *b;
}

void add_on_device(const double *a, const double *b, double *c)
{
    // Declare device pointers
    double *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void **)&d_a, sizeof(double));
    hipMalloc((void **)&d_b, sizeof(double));
    hipMalloc((void **)&d_c, sizeof(double));

    // Copy data from host to device
    hipMemcpy(d_a, a, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    add_kernel<<<1, 1>>>(d_a, d_b, d_c);

    // Copy data from device to host
    hipMemcpy(c, d_c, sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

template <typename T>
__global__ void add_arrays_kernel(T *a, T *b, int n, T *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

template __global__ void add_arrays_kernel<double>(double *a, double *b, int n, double *c);
template __global__ void add_arrays_kernel<int64_t>(int64_t *a, int64_t *b, int n, int64_t *c);

template <typename T>
void add_arrays_on_device(const T *a, const T *b, int n, T *c)
{
    // Declare device pointers
    T *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void **)&d_a, n * sizeof(T));
    hipMalloc((void **)&d_b, n * sizeof(T));
    hipMalloc((void **)&d_c, n * sizeof(T));

    // Copy data from host to device
    hipMemcpy(d_a, a, n * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(T), hipMemcpyHostToDevice);

    // Launch kernel
    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;
    add_arrays_kernel<<<num_blocks, block_size>>>(d_a, d_b, n, d_c);

    // Copy data from device to host
    hipMemcpy(c, d_c, n * sizeof(T), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

template void add_arrays_on_device<double>(const double *a, const double *b, int n, double *c);
template void add_arrays_on_device<int64_t>(const int64_t *a, const int64_t *b, int n, int64_t *c);
